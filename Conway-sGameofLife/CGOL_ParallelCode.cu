#include "hip/hip_runtime.h"
/*Name: Brijesh Mavani
CWID: A20406960
University: Illinois Institute of Technology 
Course: Parallel and Distributed Processing
Assignment: 5
*/

/*Conway's Game of Life parallel implementation using CUDA. */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <unistd.h>

int iterations,size,i,j,n;
typedef unsigned char ubyte;

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

__global__ void iterate(int n, ubyte* cBoard, ubyte* nBoard)
{
   // Assigning starting position for each thread.
	int x = blockIdx.x * 16 + threadIdx.x;
	int y = blockIdx.y * 16 + threadIdx.y;

	int k = x * n + y;
	int num = 0;
	nBoard[k] = cBoard[k];
	
	
    num+=(x-1 >= 0 && x-1 < n && y >= 0 && y <n )?cBoard[(x-1)*n+y]:0;  // check neighbor at top
	num+=(x+1 >= 0 && x+1 < n && y >= 0 && y <n )?cBoard[(x+1)*n+y]:0; // check neighbor at bottom
	num+=(x >= 0 && x < n && y-1 >= 0 && y-1 <n )?cBoard[x*n+(y-1)]:0;  // check neighbor at left
	num+=(x >= 0 && x < n && y+1 >= 0 && y+1 <n )?cBoard[x*n+(y+1)]:0;  // check neighbor at right
	num+=(x-1 >= 0 && x-1 < n && y-1 >= 0 && y-1 <n )?cBoard[(x-1)*n+(y-1)]:0;  // check neighbor at top-left
	num+=(x-1 >= 0 && x-1 < n && y+1 >= 0 && y+1 <n )?cBoard[(x-1)*n+(y+1)]:0;  // check neighbor at top-right
	num+=(x+1 >= 0 && x+1 < n && y-1 >= 0 && y-1 <n )?cBoard[(x+1)*n+(y-1)]:0;  // check neighbor at bottom-left
	num+=(x+1 >= 0 && x+1 < n && y+1 >= 0 && y+1 <n )?cBoard[(x+1)*n+(y+1)]:0;  // check neighbor at bottom-right

	// Apply game rules:
		
	//Any live cell with fewer than two live neighbors dies, as if caused by under-population	
	//Any live cell with two or three live neighbors lives on to the next generation.
	//Any live cell with more than three live neighbors dies, as if by overcrowding	
     if (num<2||num>3)
	 {
		nBoard[k] =0;
	 }	
	 
	//Any dead cell with exactly three live neighbors becomes a live cell, as if by reproduction.
	if (num==3&&!cBoard[k])
	 {
		nBoard[k] =1; 
	 }
}


void parameters(int argc, char **argv) 
{
	 /* Read command-line arguments */
	if (argc > 1)
	{
		iterations = atoi(argv[1]);
		size = atoi(argv[2]);
		printf("Interations: %d Size: %d\n",iterations,size);
	}
	else
	{
	  printf("Please provide number of interations in terms of 10/100/1000 and size for matrix.\n"); 
	}  

}
void initialize_inputs(ubyte* matrix)
{
	for (i = 0; i < size; i++)
	{
	   for (j = 0; j < size; j++)  
	   {
	     matrix[i * size + j] = rand() % 2;
	   } 
	}
}

void print_inputs(ubyte* matrix)
{
	printf("Legends: a: Alive cell, d: Dead cell.\n");
	printf("Initial Matrix:\n");
    for (i = 0; i < size; i++)
	{
		for (j = 0; j < size; j++)
		{
			printf("%s ", matrix[i * size + j]? "a|" : "d|");
		}
		printf("\n");
	}

}
int main(int argc, char* argv[])
{
 	/* Process program parameters */
	parameters(argc, argv);
    ubyte* matrix = (ubyte *)malloc(size * size * sizeof(ubyte));
	//Initialize the Matrix
	initialize_inputs(matrix);
	
	//Print the initialized Matrix
	print_inputs(matrix);
		
	printf("Running for total %d iterations.\n", iterations);
	srand(time(0));
	
	//Creating matrix in GPU to load initial data from CPU	
	ubyte* currentmatrix;
	hipMalloc((void **)&currentmatrix, size * size * sizeof(int));
	if (currentmatrix == NULL)
    {
       printf( "Memory allocation issue for current matrix.\n");
	   return false;
	}
	// Copy initial matrix to GPU currentmatrix.
	hipMemcpy(currentmatrix, matrix, size * size * sizeof(ubyte), hipMemcpyHostToDevice);
	cudaCheckErrors("Error when copying the initial matrix to the GPU.\n");

	//Creating matrix in GPU to save intermediate result between each iterations
	ubyte* nextmatrix;
	hipMalloc((void **)&nextmatrix, size * size * sizeof(ubyte));
	if (nextmatrix == NULL)
    {
       printf( "Memory allocation issue for next matrix.\n");
	   return false;
	}
	
	//initialize nextmatrix as all 0.
	hipMemset(nextmatrix, 0, size * size * sizeof(ubyte));
	cudaCheckErrors("Error when copying the next matrix to the GPU.\n");
	
	//Defining num of threads and block to execute in GPU
	dim3 threadsPerBlock(16, 16); 
	dim3 numBlocks(size/threadsPerBlock.x,size/threadsPerBlock.y);
	
	struct timeval starttime;
	gettimeofday(&starttime, NULL); //Initial time before computing starts. 

	ubyte* cmatrix;
	ubyte* nmatrix;
	int ite;
	 
	for (ite = 0; ite < iterations; ite++)
	{
		// Swap pointers every iterations to make sure next iteration uses solved matrix of previous iteration.
		if ((ite % 2) == 0)
		{
			cmatrix = currentmatrix;
			nmatrix = nextmatrix;
		}
		else
		{
			cmatrix = nextmatrix;
			nmatrix = currentmatrix;
		}

		iterate<<<numBlocks, threadsPerBlock>>>(size, cmatrix, nmatrix);
		/*if(ite <3) // for debugging the results. Checked the computed result manually to make sure code is working as expected.
		{
		  // copy the results after above mentioned iteration to CPU for printing.
		  hipMemcpy(matrix, cmatrix, size * size * sizeof(ubyte), hipMemcpyDeviceToHost);		
		  printf("Printing matrix after iterations number :%d \n",ite+1);
		  for (i = 0; i < 10; i++)
			{
				for (j = 0; j < 10; j++)
				{
					printf("%s ", matrix[i * size + j]? "a|" : "d|");
				}
				printf("\n");
			}
		}*/
		
		
		if(ite ==9||ite ==99||ite ==999)
		{
		  // copy the results after above mentioned iteration to CPU for printing.
		  hipMemcpy(matrix, cmatrix, size * size * sizeof(ubyte), hipMemcpyDeviceToHost);		
		  printf("Printing matrix after iterations number :%d \n",ite+1);
		  for (i = 0; i < 10; i++)
			{
				for (j = 0; j < 10; j++)
				{
					printf("%s ", matrix[i * size + j]? "a|" : "d|");
				}
				printf("\n");
			}
		}		
	}
	
	// copy the final result after N iterations 
	hipMemcpy(matrix, cmatrix, size * size * sizeof(ubyte), hipMemcpyDeviceToHost);
	
	struct timeval endtime;
	gettimeofday(&endtime, NULL);  //End time After computing ends. 
	double t = ((endtime.tv_sec - starttime.tv_sec) * 1000.0) + ((endtime.tv_usec - starttime.tv_usec) / 1000.0);

		
	printf("Final Matrix after %d iterations: \n",iterations);
		  for (i = 0; i < size; i++)
			{
				for (j = 0; j < size; j++)
				{
					printf("%s ", matrix[i * size + j]? "a|" : "d|");
				}
				printf("\n");
			}

	hipFree(nextmatrix);
	hipFree(currentmatrix);
	free(matrix);

	printf("%d iterations in %f milliseconds\n", iterations, t);

	return 0;
}
